#include "hip/hip_runtime.h"
#include "gpuAsciifier.cuh"
#include <math.h>


cmdplay::gpuAsciiFier::gpuAsciiFier(const std::string& brightnessLevels, int frameWidth, int frameHeight,
	bool useColors, bool useColorDithering, bool useTextDithering, bool useAccurateColors, bool useAccurateColorsFullPixel):
	m_brightnessLevels(brightnessLevels), m_frameWidth(frameWidth), m_frameHeight(frameHeight),
	m_useColorDithering(useColorDithering), m_useTextDithering(useTextDithering),
	m_brightnessLevelCount(static_cast<uint8_t>(brightnessLevels.length())), m_useColors(useColors),
	m_useAccurateColors(useAccurateColors), m_useAccurateColorsFullPixel(useAccurateColorsFullPixel)
{
	m_framepixelbytescount = m_frameWidth * m_frameHeight * 4;
	m_framebuffersize = (m_frameWidth+1) * m_frameHeight;
	m_frameWidthWithStride = m_frameWidth;
}

int cmdplay::gpuAsciiFier::getBufferSize() {
	return m_framebuffersize;
}

char* cmdplay::gpuAsciiFier::BuildFrame(uint8_t * d_rgbData) {

	int rgbsize, framecharssize, brightnesslevelsize;
	framecharssize = sizeof(char) * m_framebuffersize;
	brightnesslevelsize = sizeof(char) * m_brightnessLevelCount;

	char* framechars = (char*)malloc(sizeof(char) * framecharssize);

	char* d_framechars;
	char * d_brightnessLevels;

	hipMalloc((void**)&d_framechars, framecharssize);
	hipMalloc((void**)&d_brightnessLevels,brightnesslevelsize);

	hipMemcpy(d_brightnessLevels, m_brightnessLevels.c_str(), m_brightnessLevelCount, hipMemcpyHostToDevice);

	asciifier<< <m_frameHeight*m_frameWidth/1024 + 1, 1024>> > (d_rgbData, d_framechars,d_brightnessLevels, m_brightnessLevelCount, m_frameWidth);
	hipFree(d_brightnessLevels);

	hipMemcpy(framechars, d_framechars, framecharssize, hipMemcpyDeviceToHost);
	hipFree(d_framechars);

	for (int i = 1; i < m_framebuffersize / (m_frameWidthWithStride + 1) + 1; ++i) {
		memset(framechars + i * (m_frameWidthWithStride + 1) - 1,'\n', 1);

	}
	memset(framechars + m_framebuffersize, '\0', 1);

	return framechars;
}

__global__ void asciifier(uint8_t * rgbData, char * framechars, char * brightnesslevel, int brightnesslevelcount, int width) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int byteindex = index * 4;
	float check = static_cast<float>(index) / width;
	int frameindex = index + (int)floor(check);

	float r, g, b;
	r = (float)(int)rgbData[byteindex] / 255;
	g = (float)(int)rgbData[byteindex + 1] / 255;
	b = (float)(int)rgbData[byteindex + 2] / 255;

	int brightnessindex = (0.299 * r + 0.587 * g + 0.114 * b) * brightnesslevelcount;

	if (brightnessindex < 0) {
		brightnessindex = 0;
	}
	else if (brightnessindex >= brightnesslevelcount) {
		brightnessindex = brightnesslevelcount - 1;
	}

	framechars[frameindex] = brightnesslevel[brightnessindex];
}
#include "hip/hip_runtime.h"
#include "gpuAsciifier.cuh"
#include <math.h>


cmdplay::gpuAsciiFier::gpuAsciiFier(const std::string& brightnessLevels, int frameWidth, int frameHeight,
	bool useColors, bool useColorDithering, bool useTextDithering, bool useAccurateColors, bool useAccurateColorsFullPixel):
	m_brightnessLevels(brightnessLevels), m_frameWidth(frameWidth), m_frameHeight(frameHeight),
	m_useColorDithering(useColorDithering), m_useTextDithering(useTextDithering),
	m_brightnessLevelCount(static_cast<uint8_t>(brightnessLevels.length())), m_useColors(useColors),
	m_useAccurateColors(useAccurateColors), m_useAccurateColorsFullPixel(useAccurateColorsFullPixel)
{
	m_framepixelbytescount = m_frameWidth * m_frameHeight * 4;
	m_framebuffersize = (m_frameWidth+1) * m_frameHeight;
	m_frameWidthWithStride = m_frameWidth;
}

char* cmdplay::gpuAsciiFier::BuildFrame(uint8_t * d_rgbData) {
	char* framechars;
	//uint8_t* rgb;
	char* brightnesslevel;

	hipDeviceProp_t p;

	int rgbsize, framecharssize, brightnesslevelsize;
	//rgbsize = sizeof(unsigned char) * m_framepixelbytescount;
	framecharssize = sizeof(char) * m_framebuffersize;
	brightnesslevelsize = sizeof(char) * m_brightnessLevelCount;

	char* d_framechars;
	//uint8_t* d_rgb;
	char * d_brightnessLevels;

	hipSetDeviceFlags(hipDeviceMapHost);

	hipGetDeviceProperties(&p, 0);

	if (!p.canMapHostMemory)
		exit(0);

	hipHostAlloc((void**)&framechars, framecharssize, hipHostMallocWriteCombined | hipHostMallocMapped);
	//hipHostAlloc((void**)&rgb, rgbsize, hipHostMallocWriteCombined | hipHostMallocMapped);
	hipHostAlloc((void**)&brightnesslevel,brightnesslevelsize, hipHostMallocWriteCombined | hipHostMallocMapped);

	//memcpy(rgb, rgbData, rgbsize);

	char* h_bri = (char*)m_brightnessLevels.c_str();

	memcpy(brightnesslevel, h_bri, brightnesslevelsize);

	for (int i = 1; i < m_framebuffersize / (m_frameWidthWithStride + 1) + 1; ++i) {
		memset(framechars + i * (m_frameWidthWithStride + 1) - 1,'\n', 1);

	}
	memset(framechars + m_framebuffersize, '\0', 1);

	hipHostGetDevicePointer((void**)&d_framechars, framechars, 0);
	//hipHostGetDevicePointer((void**)&d_rgb, rgb, 0);
	hipHostGetDevicePointer((void**)&d_brightnessLevels, brightnesslevel, 0);

	asciifier<< <m_frameHeight*m_frameWidth/256 + 1, 256>> > (d_rgbData, d_framechars,d_brightnessLevels, m_brightnessLevelCount, m_frameWidth);

	hipDeviceSynchronize();

	//hipHostFree(rgb);
	hipHostFree(brightnesslevel);

	return framechars;
}

__global__ void asciifier(uint8_t * rgbData, char * framechars, char * brightnesslevel, int brightnesslevelcount, int width) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int byteindex = index * 4;
	float check = static_cast<float>(index) / width;
	int frameindex = index + (int)floor(check);

	float r, g, b;
	r = (float)(int)rgbData[byteindex] / 255;
	g = (float)(int)rgbData[byteindex + 1] / 255;
	b = (float)(int)rgbData[byteindex + 2] / 255;

	int brightnessindex = (0.299 * r + 0.587 * g + 0.114 * b) * brightnesslevelcount;

	if (brightnessindex < 0) {
		brightnessindex = 0;
	}
	else if (brightnessindex >= brightnesslevelcount) {
		brightnessindex = brightnesslevelcount - 1;
	}

	framechars[frameindex] = brightnesslevel[brightnessindex];
}
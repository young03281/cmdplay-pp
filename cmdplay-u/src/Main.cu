#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include "Version.hpp"
#include "VideoPlayer.hpp"
#include "ConsoleUtils.hpp"
#include <string>

constexpr const char* BRIGHTNESS_LEVELS = " .-+*wGHM#&%";


int main(int argc, char* argv[])
{
	std::cout << "cmdplay++ version " << cmdplay::VERSION << std::endl;
	std::string filenameInput;
	if (argc > 1)
	{
		filenameInput = std::string(argv[1]);
	}
	else
	{
		std::cout << "Enter filename: ";
		filenameInput = "D:/BB.mp4";
		if (strlen(filenameInput.c_str()) == 0)
		{
			std::cout << "Invalid filename" << std::endl;
			return EXIT_FAILURE;

		}
	}

	if (filenameInput.length() > 2) // Remove quotation marks if present
	{
		if (filenameInput[0] == '"')
			filenameInput = filenameInput.substr(1);
		if (filenameInput[filenameInput.length() - 1] == '"')
			filenameInput = filenameInput.substr(0, filenameInput.length() - 1);
	}

	cmdplay::VideoPlayer player(filenameInput, BRIGHTNESS_LEVELS);
	player.LoadVideo();
	player.Enter();

	return EXIT_SUCCESS;
}